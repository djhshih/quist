#include "hip/hip_runtime.h"
// hack to make nvcc work with gcc-4.7
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#include <cstdio>
#include <ctime>

#define _DEBUG

#include "../../util.hpp"
#include "../kernel/inclusive_scan_kernel.hpp"
#include "../functor/static_matrix_functor.hpp"


using namespace std;

typedef float real_t;


int main(int argc, char* argv[]) {
	
	std::srand( (unsigned)time(NULL) );
	
	real_t *h_x, *h_y;
	//real_t *h_block_x, *h_block_y;
	real_t *d_x, *d_y, *d_block_x, *d_block_y;
	
	// assume N < 2^22
	// since O(N) shared memory is used, the application will likely be memory-bound
	
	const size_t N = 512;
	const size_t mat_dim = 3;
	const size_t m = mat_dim * mat_dim;
	
	size_t elemPerBlock = 128;
	
	dim3 block_dim = elemPerBlock / 2;
	dim3 grid_dim = N / elemPerBlock + (N % elemPerBlock == 0 ? 0 : 1);
	
	size_t nbytes = N * sizeof(real_t) * m;
	size_t nbytes_block = grid_dim.x * sizeof(real_t) * m;
	
	// allocate array on host
	h_x = (real_t*)malloc(nbytes);
	h_y = (real_t*)malloc(nbytes);
	//h_block_x = (real_t*)malloc(nbytes_block);
	//h_block_y = (real_t*)malloc(nbytes_block);
	
	
	// allocate array on device
	hipMalloc((void**) &d_x, nbytes);
	hipMalloc((void**) &d_y, nbytes);
	hipMalloc((void**) &d_block_x, nbytes_block);
	hipMalloc((void**) &d_block_y, nbytes_block);
	
	// initialize host array
	for (size_t i = 0; i < N*m; ++i) {
		h_x[i] = (rand() % 3) - 1;
	}
	
	// copy data to device
	hipMemcpy(d_x, h_x, nbytes, hipMemcpyHostToDevice);
	
	//ScalarAdder<real_t> adder;
	//ScalarSetter<real_t> setter;
	//ArrayAdder<real_t> adder(m);
	//ArraySetter<real_t> setter(m);
	StaticMatrixMultiplierPrefixStable<real_t, mat_dim> adder;
	StaticMatrixSetter<real_t, mat_dim> setter;
	//MatrixMultiplier<real_t> adder(mat_dim);
	//MatrixSetter<real_t> setter(mat_dim);
	
	// do calculation on device
	
	// elements are divided into blocks
	// each thread processes two elements within a block
	prescan<m> <<< grid_dim, block_dim, elemPerBlock*sizeof(real_t) * m >>> (elemPerBlock, d_x, d_y, adder, setter);
	
	// one block; each thread processes a scan block from above
	aggregate_block_sum<m> <<< 1, grid_dim >>> (elemPerBlock, d_y, d_block_x, setter);
	
	// one block; each thread processes two scan block sums (hence need half the number of scan blocks from previous run)
	prescan<m> <<< 1, grid_dim.x/2, grid_dim.x*sizeof(real_t) * m >>> (grid_dim.x, d_block_x, d_block_y, adder, setter);
	
	// each thread processes one element in original data
	// need twice as many blocks as before, since each thread now processes one element
	add_block_cumsum<m> <<< grid_dim.x*2, block_dim >>> (N, d_block_y, d_y, adder, setter);

	// retrieve results from device and store it in host array
	hipMemcpy(h_y, d_y, nbytes, hipMemcpyDeviceToHost);
	//hipMemcpy(h_block_x, d_block_x, nbytes_block, hipMemcpyDeviceToHost);
	//hipMemcpy(h_block_y, d_block_y, nbytes_block, hipMemcpyDeviceToHost);
	
	// compute gold standard
	real_t* h_gold = new real_t[N*m];
	setter(h_gold[0], h_x[0]);
	for (size_t i = 1; i < N; ++i) {
		setter(h_gold[i*m], h_x[i*m]);
		adder(h_gold[i*m], h_gold[(i-1)*m]);
	}
	
	// print results
	bool equal = true;
	for (size_t i = 0; i < N*m; ++i) {
		printf("%lu %.0f %.0f %.0f", i, h_x[i], h_y[i], h_gold[i]);
		if (std::abs(h_y[i] - h_gold[i]) > 1e-5) {
			equal = false;
			printf("*\n");
		} else {
			printf("\n");
		}
	}
	if (!equal) printf("Differences detected!\n");
	
	/*
	for (size_t i = 0; i < grid_dim.x*m; ++i) {
		printf("%d %.0f %.0f\n", i, h_block_x[i], h_block_y[i]);
	}
	*/
	
	CUDA_CHECK_ERROR("end");
	
	// clean up
	free(h_x);
	free(h_y);
	//free(h_block_x);
	//free(h_block_y);
	
	delete [] h_gold;
	
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_block_x);
	hipFree(d_block_y);
	
}
