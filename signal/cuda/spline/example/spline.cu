// hack to make nvcc work with gcc-4.7
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#include <cstdio>
#include <cmath>
#include <ctime>

#include <numeric/bla.hpp>

#include "../spline.hpp"

using namespace std;

typedef double real_t;


int main(int argc, char* argv[]) {
	
	std::srand( (unsigned)time(NULL) );
	
	real_t *h_x, *h_y, *h_xx, *h_yy;
	real_t *d_x, *d_y, *d_xx, *d_yy;
	
	const size_t stride = 32;
	//const size_t N = 64;
	const size_t N = 128;
	const size_t nn = (N-1) * stride;
	
	
	size_t nbytes = N * sizeof(real_t);
	size_t nbytes_nn = nn * sizeof(real_t);
	
	// allocate array on host
	h_x = (real_t*)malloc(nbytes);
	h_y = (real_t*)malloc(nbytes);
	h_xx = (real_t*)malloc(nbytes_nn);
	h_yy = (real_t*)malloc(nbytes_nn);
	
	// allocate array on device
	hipMalloc((void**) &d_x, nbytes);
	hipMalloc((void**) &d_y, nbytes);
	hipMalloc((void**) &d_xx, nbytes_nn);
	hipMalloc((void**) &d_yy, nbytes_nn);
	
	// initialize host array
	for (size_t i = 0; i < N; ++i) {
		h_x[i] = i * stride;
		h_y[i] = (real_t)std::rand() / RAND_MAX;
	}
	for (size_t i = 0; i < nn; ++i) {
		h_xx[i] = i;
	}
	
	// clear device output arrays
	//hipMemset(d_yy, 0, nbytes_nn);
	
	// copy data to device
	hipMemcpy(d_x, h_x, nbytes, hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, nbytes, hipMemcpyHostToDevice);
	hipMemcpy(d_xx, h_xx, nbytes_nn, hipMemcpyHostToDevice);
	
	splint(N, d_x, d_y, nn, d_xx, d_yy);
	
	// retrieve results from device and store it in host array
	hipMemcpy(h_yy, d_yy, nbytes_nn, hipMemcpyDeviceToHost);
	
	// compute gold standard
	real_t* h_yy_gold = new real_t[nn];
	bla::splint(N, h_x, h_y, nn, h_xx, h_yy_gold);
	
	// print results
	for (size_t i = 0; i < nn; ++i) {
		printf("%lu %f %f\n", i, h_yy[i], h_yy_gold[i]);
	}
	
	// clean up
	free(h_x);
	free(h_y);
	free(h_xx);
	free(h_yy);
	delete [] h_yy_gold;
	
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_xx);
	hipFree(d_yy);
	
	CUDA_CHECK_ERROR("main");
}
