#include "hip/hip_runtime.h"
// hack to make nvcc work with gcc-4.7
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#include <cstdio>
#include <cmath>

#include "emd_kernel.hpp"
#include "../emd.hpp"

using namespace std;


int main(int argc, char* argv[]) {

	float *h_x, *h_y, *h_modes;
	unsigned *h_counts;
	float *d_x, *d_y, *d_modes;
	unsigned *d_counts;
	
	const size_t N = 32 * 32;
	const size_t ns = 32 * 32 / 4;
	const size_t nr = 1024;
	// note: when the window size is too small (i.e. does not contain enough extrema), IMF cannot be extracted
	
	//size_t k = log2((float)N) + 1;
	size_t k = 4;
	
	dim3 block_dim = 1;
	//dim3 grid_dim = N / wsize / block_dim.x + (N%block_dim.x == 0 ? 0 : 1);
	dim3 grid_dim = nr;
	
	size_t nbytes = N * sizeof(float);
	size_t nbytes_modes = k * N * sizeof(float);
	size_t nbytes_counts = N * sizeof(unsigned);
	
	// allocate array on host
	h_x = (float*)malloc(nbytes);
	h_y = (float*)malloc(nbytes);
	h_modes = (float*)malloc(nbytes_modes);
	h_counts = (unsigned*)malloc(nbytes_counts);
	

	// allocate array on device
	hipMalloc((void**) &d_x, nbytes);
	hipMalloc((void**) &d_y, nbytes);
	hipMalloc((void**) &d_modes, nbytes_modes);
	hipMalloc((void**) &d_counts, nbytes_counts);

	// initialize host array
	for (size_t i = 0; i < N; i++) {
		float x = (float)i/M_PI;
		h_x[i] = x;
		h_y[i] = sin(x) + 0.5 * sin(x/10);
	}
	
	// clear device output arrays
	hipMemset(d_modes, 0, nbytes_modes);
	hipMemset(d_counts, 0, nbytes_counts);
	
	hipDeviceSynchronize();
	
	// copy data to device
	hipMemcpy(d_x, h_x, nbytes, hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, nbytes, hipMemcpyHostToDevice);

	// do calculate on device
	dsemd <<< grid_dim, block_dim >>> (N, d_x, d_y, ns, nr, d_counts, k, d_modes);
	//scale <<< N, 1 >>> (N, k, d_modes, d_counts);

	// retrieve results from device and store it in host array
	hipMemcpy(h_modes, d_modes, nbytes_modes, hipMemcpyDeviceToHost);
	hipMemcpy(h_counts, d_counts, nbytes_counts, hipMemcpyDeviceToHost);
	
	// scale the ensemble values
	for (size_t i = 0; i < k; ++i) {
		for (size_t j = 0; j < N; ++j) {
			h_modes[i*N + j] /= h_counts[j];
		}
	}
	
	// compute gold standard
	float** gold_modes = emd::emd(N, h_x, h_y, &k);
	//float** gold_modes = emd::dsemd(N, h_x, h_y, &k, ns, nr);

	// print results
	for (size_t i = 0; i < k; ++i) {
		for (size_t j = 0; j < N; ++j) {
			printf("%d %d %d %f %f\n", i, j, h_counts[j], h_modes[i*N + j], gold_modes[i][j]);
		}
	}
	
	hipError_t code = hipGetLastError();
	if (code != hipSuccess) {
		printf("%s\n", hipGetErrorString(code));
		printf("grid_dim = %d, block_dim = %d\n", grid_dim.x, block_dim.x);
	}

	// clean up
	free(h_x);
	free(h_y);
	free(h_modes);
	free(h_counts);
	
	emd::free_arrays(gold_modes, k);
	
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_modes);
	hipFree(d_counts);
}
